#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

// Write down the kernels here

//It calculate the number of requests for a facility of a center.
__global__ void req_per_facility(int *d_req_cen, int *d_req_fac, int *d_start_cen, int *d_req_per_fac, int size){
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < size){
    int index = d_start_cen[d_req_cen[id]] + d_req_fac[id];
    atomicAdd(&d_req_per_fac[index],1);

  }
}


//Find number of success request per center.

__global__ void success_request(int *d_start_fac, int *d_fac_id_req, int *d_req_start, int *d_req_slots, int *d_capacity, int *d_success, int *d_req_cen, int size, int *total_succ){
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < size){
    int time[24]; // It is storing the number less than capacity of request at a time.
    for(int i=0;i<24;i++){
      time[i]=0;
    }
    
    int i = d_start_fac[id];
    while(i<d_start_fac[id+1]){
      int st_time = d_req_start[d_fac_id_req[i]]-1; // starting time of request.
      int end_time = st_time +d_req_slots[d_fac_id_req[i]]; // end time of request.
      int flag = true;
      for(int j=st_time ; j< end_time; j++){
        if(d_capacity[id] <= time[j]){ //if time value of j is greater than capacity of request than set flag equal to false.
          flag=false;
        }
      }
      if(flag){
        for(int j=st_time; j< end_time ; j++){
          time[j]++;
        }
        atomicAdd(&d_success[d_req_cen[d_fac_id_req[i]]],1); //Increase the given success of center.
        atomicAdd(&total_succ[0],1);
      }
      i++;
    }
  }
}

//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		
    //------------------------cpu code-------------------------------//
    
    int *start_cen, *req_per_fac, *start_fac, *fac_id_req, *id_inc;
    int *d_capacity, *d_req_id, *d_req_cen, *d_req_fac, *d_req_per_fac, *d_start_cen, *d_start_fac, *d_id_inc, *d_fac_id_req, *d_success;
    int *d_req_start, *d_req_slots;

    
    start_cen = (int *)malloc(N*sizeof(int));//It is use to store the starting index of each center.

    //calculate the starting index of facility.
    int sum=0;
    for(int i=0;i<N;i++){
      start_cen[i]=sum;
      sum+=facility[i];
    }

    id_inc = (int *)malloc((sum+1)*sizeof(int));
    req_per_fac = (int *)malloc(sum*sizeof(int)); //it use to store the number of requests per facility.
    start_fac = (int *)malloc((sum+1)*sizeof(int)); //Store staring index of each facility request.
    fac_id_req = (int *)malloc(R*sizeof(int));  //Store the req_id of the facility request.

    //Memory allocation on GPU.
    hipMalloc(&d_start_cen,N*sizeof(int));
    hipMalloc(&d_start_fac,(sum+1)*sizeof(int));
    hipMalloc(&d_req_per_fac,sum*sizeof(int));
    hipMalloc(&d_capacity,sum*sizeof(int));
    hipMalloc(&d_req_id, R*sizeof(int));
    hipMalloc(&d_req_cen,R*sizeof(int));
    hipMalloc(&d_req_fac,R*sizeof(int));
    hipMalloc(&d_req_start,R*sizeof(int));
    hipMalloc(&d_req_slots,R*sizeof(int));
    hipMalloc(&d_start_fac,(sum + 1)*sizeof(int));
    hipMalloc(&d_success,N*sizeof(int));
    hipMalloc(&d_id_inc,(sum+1)*sizeof(int));
    
    //Set value to 0.
    hipMemset(d_req_per_fac , 0 ,sum*sizeof(int));
    hipMemset(d_success, 0 , N*sum*sizeof(int));

    //Memory copy from CPU to GPU.
    hipMemcpy(d_start_cen, start_cen, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_capacity, capacity, sum*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_id, req_id, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_cen, req_cen, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_fac, req_fac, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_start, req_start, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_slots, req_slots, R*sizeof(int), hipMemcpyHostToDevice);
    
    //Kernel call to calculate the no. of request per facility.
    int no_of_blocks = ceil((float)R/1024);
    req_per_facility<<< no_of_blocks, 1024>>>(d_req_cen, d_req_fac, d_start_cen, d_req_per_fac, R);
    hipMemcpy(req_per_fac, d_req_per_fac, sum*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //calculate the starting index of each facility.
    int fac_sum=0;
    for(int i=0; i<sum+1; i++){
      start_fac[i]=fac_sum;
      id_inc[i]=fac_sum;
      fac_sum += req_per_fac[i];
    }


    hipMalloc(&d_fac_id_req, R*sizeof(int));
    hipMemcpy(d_start_fac, start_fac, (sum+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_id_inc, start_fac, (sum+1)*sizeof(int), hipMemcpyHostToDevice);
    
    //index maping to facility.
    for(int i=0; i<R; i++){
      int index = start_cen[req_cen[i]] + req_fac[i];
      fac_id_req[id_inc[index]] = req_id[i];
      id_inc[index]++;
    }


    hipMemcpy(d_fac_id_req, fac_id_req , R*sizeof(int), hipMemcpyHostToDevice);

  
    //*********************************
    // Call the kernels here

    //Calculate the success request at centers.
    int *total_succ;
    hipMalloc(&total_succ,sizeof(int));
    hipMemset(total_succ, 0, sizeof(int));
    no_of_blocks = ceil((float)sum/1024);
    success_request<<<no_of_blocks, 1024>>>(d_start_fac, d_fac_id_req, d_req_start, d_req_slots, d_capacity, d_success, d_req_cen, sum, total_succ);
    hipMemcpy(succ_reqs, d_success, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&success, total_succ, sizeof(int), hipMemcpyDeviceToHost);
    //********************************

    fail = R-success;



    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}